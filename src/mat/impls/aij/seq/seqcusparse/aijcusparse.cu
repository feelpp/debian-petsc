/*
  Defines the basic matrix operations for the AIJ (compressed row)
  matrix storage format using the CUSPARSE library,
*/

#include "petscconf.h"
#include "../src/mat/impls/aij/seq/aij.h"          /*I "petscmat.h" I*/
#include <../src/mat/impls/sbaij/seq/sbaij.h>
#include "../src/vec/vec/impls/dvecimpl.h"
#include "petsc-private/vecimpl.h"
#undef VecType
#include "cusparsematimpl.h"

const char *const MatCUSPARSEStorageFormats[] = {"CSR","ELL","HYB","MatCUSPARSEStorageFormat","MAT_CUSPARSE_",0};

/* this is such a hack ... but I don't know of another way to pass this variable
   from one GPU_Matrix_Ifc class to another. This is necessary for the parallel
   SpMV. Essentially, I need to use the same stream variable in two different
   data structures. I do this by creating a single instance of that stream
   and reuse it. */
hipStream_t theBodyStream=0;

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(Mat);
static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);

#undef __FUNCT__
#define __FUNCT__ "MatFactorGetSolverPackage_seqaij_cusparse"
PetscErrorCode MatFactorGetSolverPackage_seqaij_cusparse(Mat A,const MatSolverPackage *type)
{
  PetscFunctionBegin;
  *type = MATSOLVERCUSPARSE;
  PetscFunctionReturn(0);
}

/*MC
  MATSOLVERCUSPARSE = "cusparse" - A matrix type providing triangular solvers for seq matrices
  on a single GPU of type, seqaijcusparse, aijcusparse, or seqaijcusp, aijcusp. Currently supported
  algorithms are ILU(k) and ICC(k). Typically, deeper factorizations (larger k) results in poorer
  performance in the triangular solves. Full LU, and Cholesky decompositions can be solved through the
  CUSPARSE triangular solve algorithm. However, the performance can be quite poor and thus these
  algorithms are not recommended. This class does NOT support direct solver operations.

  ./configure --download-txpetscgpu to install PETSc to use CUSPARSE

  Consult CUSPARSE documentation for more information about the matrix storage formats
  which correspond to the options database keys below.

   Options Database Keys:
.  -mat_cusparse_solve_storage_format csr - sets the storage format matrices (for factors in MatSolve) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid). Only available with 'txpetscgpu' package.

  Level: beginner

.seealso: PCFactorSetMatSolverPackage(), MatSolverPackage, MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

#undef __FUNCT__
#define __FUNCT__ "MatGetFactor_seqaij_cusparse"
PETSC_EXTERN PetscErrorCode MatGetFactor_seqaij_cusparse(Mat A,MatFactorType ftype,Mat *B)
{
  PetscErrorCode ierr;
  PetscInt       n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),B);CHKERRQ(ierr);
  ierr = MatSetSizes(*B,n,n,n,n);CHKERRQ(ierr);
  ierr = MatSetType(*B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU || ftype == MAT_FACTOR_ILUDT) {
    ierr = MatSetBlockSizes(*B,A->rmap->bs,A->cmap->bs);CHKERRQ(ierr);
    (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqAIJCUSPARSE;
  } else if (ftype == MAT_FACTOR_CHOLESKY || ftype == MAT_FACTOR_ICC) {
    (*B)->ops->iccfactorsymbolic      = MatICCFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqAIJCUSPARSE;
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Factor type not supported for CUSPARSE Matrix Types");

  ierr = MatSeqAIJSetPreallocation(*B,MAT_SKIP_ALLOCATION,NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)(*B),"MatFactorGetSolverPackage_C",MatFactorGetSolverPackage_seqaij_cusparse);CHKERRQ(ierr);
  (*B)->factortype = ftype;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetFormat_SeqAIJCUSPARSE"
PETSC_INTERN PetscErrorCode MatCUSPARSESetFormat_SeqAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSPARSE_MULT:
    cusparseMat->format = format;
    break;
  case MAT_CUSPARSE_SOLVE:
    cusparseMatSolveStorageFormat = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparseMat->format           = format;
    cusparseMatSolveStorageFormat = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. MAT_CUSPARSE_MULT, MAT_CUSPARSE_SOLVE, and MAT_CUSPARSE_ALL are currently supported.",op);
  }
  PetscFunctionReturn(0);
}

/*@
   MatCUSPARSESetFormat - Sets the storage format of CUSPARSE matrices for a particular
   operation. Only the MatMult operation can use different GPU storage formats
   for MPIAIJCUSPARSE matrices. This requires the txpetscgpu package. Use --download-txpetscgpu
   to build/install PETSc to use this package.

   Not Collective

   Input Parameters:
+  A - Matrix of type SEQAIJCUSPARSE
.  op - MatCUSPARSEFormatOperation. SEQAIJCUSPARSE matrices support MAT_CUSPARSE_MULT, MAT_CUSPARSE_SOLVE, and MAT_CUSPARSE_ALL. MPIAIJCUSPARSE matrices support MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_OFFDIAG, and MAT_CUSPARSE_ALL.
-  format - MatCUSPARSEStorageFormat (one of MAT_CUSPARSE_CSR, MAT_CUSPARSE_ELL, MAT_CUSPARSE_HYB)

   Output Parameter:

   Level: intermediate

.seealso: MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
@*/
#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetFormat"
PetscErrorCode MatCUSPARSESetFormat(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  ierr = PetscTryMethod(A, "MatCUSPARSESetFormat_C",(Mat,MatCUSPARSEFormatOperation,MatCUSPARSEStorageFormat),(A,op,format));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSetFromOptions_SeqAIJCUSPARSE"
static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode           ierr;
  MatCUSPARSEStorageFormat format;
  PetscBool                flg;

  PetscFunctionBegin;
  ierr = PetscOptionsHead("SeqAIJCUSPARSE options");CHKERRQ(ierr);
  ierr = PetscObjectOptionsBegin((PetscObject)A);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)MAT_CUSPARSE_CSR,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT,format);CHKERRQ(ierr);
    }
  } else {
    ierr = PetscOptionsEnum("-mat_cusparse_solve_storage_format","sets storage format of (seq)aijcusparse gpu matrices for TriSolve",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)MAT_CUSPARSE_CSR,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_SOLVE,format);CHKERRQ(ierr);
    }
  }
  ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV and TriSolve",
                          "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)MAT_CUSPARSE_CSR,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format);CHKERRQ(ierr);
  }
  ierr = PetscOptionsEnd();CHKERRQ(ierr);
  PetscFunctionReturn(0);

}

#undef __FUNCT__
#define __FUNCT__ "MatILUFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatILUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatLUFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatLUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatICCFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatICCFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCholeskyFactorSymbolic_SeqAIJCUSPARSE"
static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCholeskyFactorSymbolic_SeqAIJ(B,A,perm,info);CHKERRQ(ierr);
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildILULowerTriMatrix"
static PetscErrorCode MatSeqAIJCUSPARSEBuildILULowerTriMatrix(Mat A)
{
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  PetscInt                     n                   = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMat        = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  hipsparseStatus_t             stat;
  const PetscInt               *ai = a->i,*aj = a->j,*vi;
  const MatScalar              *aa = a->a,*v;
  PetscInt                     *AiLo, *AjLo;
  PetscScalar                  *AALo;
  PetscInt                     i,nz, nzLower, offset, rowOffset;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    try {
      /* first figure out the number of nonzeros in the lower triangular matrix including 1's on the diagonal. */
      nzLower=n+ai[n]-ai[1];

      /* Allocate Space for the lower triangular matrix */
      ierr = hipHostMalloc((void**) &AiLo, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AjLo, nzLower*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AALo, nzLower*sizeof(PetscScalar));CHKERRCUSP(ierr);

      /* Fill the lower triangular matrix */
      AiLo[0]  = (PetscInt) 0;
      AiLo[n]  = nzLower;
      AjLo[0]  = (PetscInt) 0;
      AALo[0]  = (MatScalar) 1.0;
      v        = aa;
      vi       = aj;
      offset   = 1;
      rowOffset= 1;
      for (i=1; i<n; i++) {
        nz = ai[i+1] - ai[i];
        /* additional 1 for the term on the diagonal */
        AiLo[i]    = rowOffset;
        rowOffset += nz+1;

        ierr = PetscMemcpy(&(AjLo[offset]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
        ierr = PetscMemcpy(&(AALo[offset]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);

        offset      += nz;
        AjLo[offset] = (PetscInt) i;
        AALo[offset] = (MatScalar) 1.0;
        offset      += 1;

        v  += nz;
        vi += nz;
      }
      cusparseMat = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseTriFactors->format]);

      stat = cusparseMat->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_TRIANGULAR, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUSP(stat);
      ierr = cusparseMat->setMatrix(n, n, nzLower, AiLo, AjLo, AALo);CHKERRCUSP(ierr);
      stat = cusparseMat->solveAnalysis();CHKERRCUSP(stat);

      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = cusparseMat;

      ierr = hipHostFree(AiLo);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjLo);CHKERRCUSP(ierr);
      ierr = hipHostFree(AALo);CHKERRCUSP(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildILUUpperTriMatrix"
static PetscErrorCode MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(Mat A)
{
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  PetscInt                     n                   = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMat        = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t             stat;
  const PetscInt               *aj = a->j,*adiag = a->diag,*vi;
  const MatScalar              *aa = a->a,*v;
  PetscInt                     *AiUp, *AjUp;
  PetscScalar                  *AAUp;
  PetscInt                     i,nz, nzUpper, offset;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    try {
      /* next, figure out the number of nonzeros in the upper triangular matrix. */
      nzUpper = adiag[0]-adiag[n];

      /* Allocate Space for the upper triangular matrix */
      ierr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUSP(ierr);

      /* Fill the upper triangular matrix */
      AiUp[0]=(PetscInt) 0;
      AiUp[n]=nzUpper;
      offset = nzUpper;
      for (i=n-1; i>=0; i--) {
        v  = aa + adiag[i+1] + 1;
        vi = aj + adiag[i+1] + 1;

        /* number of elements NOT on the diagonal */
        nz = adiag[i] - adiag[i+1]-1;

        /* decrement the offset */
        offset -= (nz+1);

        /* first, set the diagonal elements */
        AjUp[offset] = (PetscInt) i;
        AAUp[offset] = 1./v[nz];
        AiUp[i]      = AiUp[i+1] - (nz+1);

        ierr = PetscMemcpy(&(AjUp[offset+1]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
        ierr = PetscMemcpy(&(AAUp[offset+1]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
      }
      cusparseMat = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseTriFactors->format]);

      stat = cusparseMat->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_TRIANGULAR, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
      ierr = cusparseMat->setMatrix(n, n, nzUpper, AiUp, AjUp, AAUp);CHKERRCUSP(ierr);
      stat = cusparseMat->solveAnalysis();CHKERRCUSP(stat);

      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = cusparseMat;

      ierr = hipHostFree(AiUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AAUp);CHKERRCUSP(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU"
static PetscErrorCode MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           isrow = a->row,iscol = a->icol;
  PetscBool                    row_identity,col_identity;
  const PetscInt               *r,*c;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSEBuildILULowerTriMatrix(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(A);CHKERRQ(ierr);

  cusparseTriFactors->tempvec = new CUSPARRAY;
  cusparseTriFactors->tempvec->resize(n);

  A->valid_GPU_matrix = PETSC_CUSP_BOTH;
  /*lower triangular indices */
  ierr = ISGetIndices(isrow,&r);CHKERRQ(ierr);
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  if (!row_identity) {
    ierr = cusparseTriFactors->loTriFactorPtr->setOrdIndices(r, n);CHKERRCUSP(ierr);
  }
  ierr = ISRestoreIndices(isrow,&r);CHKERRQ(ierr);

  /*upper triangular indices */
  ierr = ISGetIndices(iscol,&c);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (!col_identity) {
    ierr = cusparseTriFactors->upTriFactorPtr->setOrdIndices(c, n);CHKERRCUSP(ierr);
  }
  ierr = ISRestoreIndices(iscol,&c);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildICCTriMatrices"
static PetscErrorCode MatSeqAIJCUSPARSEBuildICCTriMatrices(Mat A)
{
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMatLo      = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc               *cusparseMatUp      = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t             stat;
  PetscErrorCode               ierr;
  PetscInt                     *AiUp, *AjUp;
  PetscScalar                  *AAUp;
  PetscScalar                  *AALo;
  PetscInt                     nzUpper = a->nz,n = A->rmap->n,i,offset,nz,j;
  Mat_SeqSBAIJ                 *b = (Mat_SeqSBAIJ*)A->data;
  const PetscInt               *ai = b->i,*aj = b->j,*vj;
  const MatScalar              *aa = b->a,*v;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    try {
      /* Allocate Space for the upper triangular matrix */
      ierr = hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void**) &AALo, nzUpper*sizeof(PetscScalar));CHKERRCUSP(ierr);

      /* Fill the upper triangular matrix */
      AiUp[0]=(PetscInt) 0;
      AiUp[n]=nzUpper;
      offset = 0;
      for (i=0; i<n; i++) {
        /* set the pointers */
        v  = aa + ai[i];
        vj = aj + ai[i];
        nz = ai[i+1] - ai[i] - 1; /* exclude diag[i] */

        /* first, set the diagonal elements */
        AjUp[offset] = (PetscInt) i;
        AAUp[offset] = 1.0/v[nz];
        AiUp[i]      = offset;
        AALo[offset] = 1.0/v[nz];

        offset+=1;
        if (nz>0) {
          ierr = PetscMemcpy(&(AjUp[offset]), vj, nz*sizeof(PetscInt));CHKERRQ(ierr);
          ierr = PetscMemcpy(&(AAUp[offset]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
          for (j=offset; j<offset+nz; j++) {
            AAUp[j] = -AAUp[j];
            AALo[j] = AAUp[j]/v[nz];
          }
          offset+=nz;
        }
      }

      /* Build the upper triangular piece */
      cusparseMatUp = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseTriFactors->format]);
      stat = cusparseMatUp->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_TRIANGULAR, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUSP(stat);
      ierr = cusparseMatUp->setMatrix(A->rmap->n, A->cmap->n, a->nz, AiUp, AjUp, AAUp);CHKERRCUSP(ierr);
      stat = cusparseMatUp->solveAnalysis();CHKERRCUSP(stat);
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = cusparseMatUp;

      /* Build the lower triangular piece */
      cusparseMatLo = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseTriFactors->format]);
      stat = cusparseMatLo->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_TRIANGULAR, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
      ierr = cusparseMatLo->setMatrix(A->rmap->n, A->cmap->n, a->nz, AiUp, AjUp, AALo);CHKERRCUSP(ierr);
      stat = cusparseMatLo->solveAnalysis(HIPSPARSE_OPERATION_TRANSPOSE);CHKERRCUSP(stat);
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = cusparseMatLo;

      /* set this flag ... for performance logging */
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->isSymmOrHerm = PETSC_TRUE;

      A->valid_GPU_matrix = PETSC_CUSP_BOTH;
      ierr = hipHostFree(AiUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AAUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AALo);CHKERRCUSP(ierr);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU"
static PetscErrorCode MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode               ierr;
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           ip = a->row;
  const PetscInt               *rip;
  PetscBool                    perm_identity;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  ierr = MatSeqAIJCUSPARSEBuildICCTriMatrices(A);CHKERRQ(ierr);
  cusparseTriFactors->tempvec = new CUSPARRAY;
  cusparseTriFactors->tempvec->resize(n);
  /*lower triangular indices */
  ierr = ISGetIndices(ip,&rip);CHKERRQ(ierr);
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (!perm_identity) {
    ierr = cusparseTriFactors->loTriFactorPtr->setOrdIndices(rip, n);CHKERRCUSP(ierr);
    ierr = cusparseTriFactors->upTriFactorPtr->setOrdIndices(rip, n);CHKERRCUSP(ierr);
  }
  ierr = ISRestoreIndices(ip,&rip);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatLUFactorNumeric_SeqAIJCUSPARSE"
static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             isrow = b->row,iscol = b->col;
  PetscBool      row_identity,col_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatLUFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);
  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (row_identity && col_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCholeskyFactorNumeric_SeqAIJCUSPARSE"
static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             ip = b->row;
  PetscBool      perm_identity;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCholeskyFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);

  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(ip,&perm_identity);CHKERRQ(ierr);
  if (perm_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
  }

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEAnalyzeTransposeForSolve"
static PetscErrorCode MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(Mat A)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMatLo      = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc               *cusparseMatUp      = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t             stat;

  PetscFunctionBegin;
  stat = cusparseMatLo->initializeCusparseMatTranspose(MAT_cusparseHandle,
                                                       HIPSPARSE_MATRIX_TYPE_TRIANGULAR,
                                                       HIPSPARSE_FILL_MODE_UPPER,
                                                       HIPSPARSE_DIAG_TYPE_UNIT);CHKERRCUSP(stat);
  stat = cusparseMatLo->solveAnalysisTranspose();CHKERRCUSP(stat);

  stat = cusparseMatUp->initializeCusparseMatTranspose(MAT_cusparseHandle,
                                                       HIPSPARSE_MATRIX_TYPE_TRIANGULAR,
                                                       HIPSPARSE_FILL_MODE_LOWER,
                                                       HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
  stat = cusparseMatUp->solveAnalysisTranspose();CHKERRCUSP(stat);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSEGenerateTransposeForMult"
static PetscErrorCode MatSeqAIJCUSPARSEGenerateTransposeForMult(Mat A)
{
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  hipsparseStatus_t   stat;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (cusparseMat->isSymmOrHerm) {
    stat = cusparseMat->mat->initializeCusparseMatTranspose(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_SYMMETRIC, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
  } else {
    stat = cusparseMat->mat->initializeCusparseMatTranspose(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
  }
  ierr = cusparseMat->mat->setMatrixTranspose(A->rmap->n, A->cmap->n, a->nz, a->i, a->j, a->a);CHKERRCUSP(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolveTranspose_SeqAIJCUSPARSE"
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  CUSPARRAY                    *xGPU, *bGPU;
  hipsparseStatus_t             stat;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMatLo      = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc               *cusparseMatUp      = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  CUSPARRAY                    *tempGPU            = (CUSPARRAY*) cusparseTriFactors->tempvec;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  /* Analyze the matrix ... on the fly */
  if (!cusparseTriFactors->hasTranspose) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    cusparseTriFactors->hasTranspose=PETSC_TRUE;
  }

  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* solve with reordering */
  ierr = cusparseMatUp->reorderIn(xGPU, bGPU);CHKERRCUSP(ierr);
  stat = cusparseMatUp->solveTranspose(xGPU, tempGPU);CHKERRCUSP(stat);
  stat = cusparseMatLo->solveTranspose(tempGPU, xGPU);CHKERRCUSP(stat);
  ierr = cusparseMatLo->reorderOut(xGPU);CHKERRCUSP(ierr);

  /* restore */
  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);

  if (cusparseTriFactors->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 3.0*A->cmap->n);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz - A->cmap->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering"
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  CUSPARRAY                    *xGPU,*bGPU;
  hipsparseStatus_t             stat;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMatLo      = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc               *cusparseMatUp      = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  CUSPARRAY                    *tempGPU            = (CUSPARRAY*) cusparseTriFactors->tempvec;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  /* Analyze the matrix ... on the fly */
  if (!cusparseTriFactors->hasTranspose) {
    ierr = MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A);CHKERRQ(ierr);
    cusparseTriFactors->hasTranspose=PETSC_TRUE;
  }

  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* solve */
  stat = cusparseMatUp->solveTranspose(bGPU, tempGPU);CHKERRCUSP(stat);
  stat = cusparseMatLo->solveTranspose(tempGPU, xGPU);CHKERRCUSP(stat);

  /* restore */
  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  if (cusparseTriFactors->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 3.0*A->cmap->n);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz - A->cmap->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolve_SeqAIJCUSPARSE"
static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  CUSPARRAY                    *xGPU,*bGPU;
  hipsparseStatus_t             stat;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMatLo      = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc               *cusparseMatUp      = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  CUSPARRAY                    *tempGPU            = (CUSPARRAY*)cusparseTriFactors->tempvec;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* solve with reordering */
  ierr = cusparseMatLo->reorderIn(xGPU, bGPU);CHKERRCUSP(ierr);
  stat = cusparseMatLo->solve(xGPU, tempGPU);CHKERRCUSP(stat);
  stat = cusparseMatUp->solve(tempGPU, xGPU);CHKERRCUSP(stat);
  ierr = cusparseMatUp->reorderOut(xGPU);CHKERRCUSP(ierr);

  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  if (cusparseTriFactors->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 3.0*A->cmap->n);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz - A->cmap->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSolve_SeqAIJCUSPARSE_NaturalOrdering"
static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  CUSPARRAY                    *xGPU,*bGPU;
  hipsparseStatus_t             stat;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc               *cusparseMatLo      = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc               *cusparseMatUp      = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  CUSPARRAY                    *tempGPU            = (CUSPARRAY*)cusparseTriFactors->tempvec;
  PetscErrorCode               ierr;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* solve */
  stat = cusparseMatLo->solve(bGPU, tempGPU);CHKERRCUSP(stat);
  stat = cusparseMatUp->solve(tempGPU, xGPU);CHKERRCUSP(stat);

  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  if (cusparseTriFactors->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 3.0*A->cmap->n);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz - A->cmap->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSECopyToGPU"
static PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat A)
{

  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  PetscInt           m = A->rmap->n,*ii,*ridx;
  PetscBool          symmetryTest=PETSC_FALSE, hermitianTest=PETSC_FALSE;
  PetscBool          symmetryOptionIsSet=PETSC_FALSE, symmetryOptionTest=PETSC_FALSE;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU) {
    ierr = PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
    /*
      It may be possible to reuse nonzero structure with new matrix values but
      for simplicity and insured correctness we delete and build a new matrix on
      the GPU. Likely a very small performance hit.
    */
    if (cusparseMat->mat) {
      try {
        delete cusparseMat->mat;
        if (cusparseMat->tempvec) delete cusparseMat->tempvec;

      } catch(char *ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
      }
    }
    try {
      cusparseMat->nonzerorow=0;
      for (int j = 0; j<m; j++) cusparseMat->nonzerorow += ((a->i[j+1]-a->i[j])>0);

      if (a->compressedrow.use) {
        m    = a->compressedrow.nrows;
        ii   = a->compressedrow.i;
        ridx = a->compressedrow.rindex;
      } else {
        /* Forcing compressed row on the GPU ... only relevant for CSR storage */
        int k=0;
        ierr = PetscMalloc((cusparseMat->nonzerorow+1)*sizeof(PetscInt), &ii);CHKERRQ(ierr);
        ierr = PetscMalloc((cusparseMat->nonzerorow)*sizeof(PetscInt), &ridx);CHKERRQ(ierr);
        ii[0]=0;
        for (int j = 0; j<m; j++) {
          if ((a->i[j+1]-a->i[j])>0) {
            ii[k]  = a->i[j];
            ridx[k]= j;
            k++;
          }
        }
        ii[cusparseMat->nonzerorow] = a->nz;

        m = cusparseMat->nonzerorow;
      }

      /* Build our matrix ... first determine the GPU storage type */
      cusparseMat->mat = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseMat->format]);

      /* Create the streams and events (if desired).  */
      PetscMPIInt size;
      ierr = MPI_Comm_size(PetscObjectComm((PetscObject)A),&size);CHKERRQ(ierr);
      ierr = cusparseMat->mat->buildStreamsAndEvents(size, &theBodyStream);CHKERRCUSP(ierr);

      ierr = MatIsSymmetricKnown(A,&symmetryOptionIsSet,&symmetryOptionTest);CHKERRQ(ierr);
      if ((symmetryOptionIsSet && !symmetryOptionTest) || !symmetryOptionIsSet) {
	/* HIPSPARSE_FILL_MODE_UPPER and HIPSPARSE_DIAG_TYPE_NON_UNIT are irrelevant here */
        hipsparseStatus_t stat = cusparseMat->mat->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
        cusparseMat->isSymmOrHerm = PETSC_FALSE;
      } else {
        ierr = MatIsSymmetric(A,0.0,&symmetryTest);CHKERRQ(ierr);
        if (symmetryTest) {
          hipsparseStatus_t stat = cusparseMat->mat->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_SYMMETRIC, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
          cusparseMat->isSymmOrHerm = PETSC_TRUE;
        } else {
          ierr = MatIsHermitian(A,0.0,&hermitianTest);CHKERRQ(ierr);
          if (hermitianTest) {
            hipsparseStatus_t stat = cusparseMat->mat->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_HERMITIAN, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
            cusparseMat->isSymmOrHerm = PETSC_TRUE;
          } else {
            /* HIPSPARSE_FILL_MODE_UPPER and HIPSPARSE_DIAG_TYPE_NON_UNIT are irrelevant here */
            hipsparseStatus_t stat = cusparseMat->mat->initializeCusparseMat(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);CHKERRCUSP(stat);
            cusparseMat->isSymmOrHerm = PETSC_FALSE;
          }
        }
      }

      /* lastly, build the matrix */
      ierr = cusparseMat->mat->setMatrix(m, A->cmap->n, a->nz, ii, a->j, a->a);CHKERRCUSP(ierr);
      cusparseMat->mat->setCPRowIndices(ridx, m);
      if (!a->compressedrow.use) {
        ierr = PetscFree(ii);CHKERRQ(ierr);
        ierr = PetscFree(ridx);CHKERRQ(ierr);
      }
      cusparseMat->tempvec = new CUSPARRAY;
      cusparseMat->tempvec->resize(m);
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
    ierr = WaitForGPU();CHKERRCUSP(ierr);

    A->valid_GPU_matrix = PETSC_CUSP_BOTH;

    ierr = PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatGetVecs_SeqAIJCUSPARSE"
static PetscErrorCode MatGetVecs_SeqAIJCUSPARSE(Mat mat, Vec *right, Vec *left)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (right) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),right);CHKERRQ(ierr);
    ierr = VecSetSizes(*right,mat->cmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*right,mat->rmap->bs);CHKERRQ(ierr);
    ierr = VecSetType(*right,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->cmap,&(*right)->map);CHKERRQ(ierr);
  }
  if (left) {
    ierr = VecCreate(PetscObjectComm((PetscObject)mat),left);CHKERRQ(ierr);
    ierr = VecSetSizes(*left,mat->rmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*left,mat->rmap->bs);CHKERRQ(ierr);
    ierr = VecSetType(*left,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->rmap,&(*left)->map);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqAIJCUSPARSE"
static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CUSPARRAY          *xarray,*yarray;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(yy,&yarray);CHKERRQ(ierr);
  try {
    ierr = cusparseMat->mat->multiply(xarray, yarray);CHKERRCUSP(ierr);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cusparseMat->mat->hasNonZeroStream()) {
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  }
  if (cusparseMat->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 3.0*cusparseMat->nonzerorow);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz - cusparseMat->nonzerorow);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMultTranspose_SeqAIJCUSPARSE"
static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CUSPARRAY          *xarray,*yarray;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  if (!cusparseMat->hasTranspose) {
    ierr = MatSeqAIJCUSPARSEGenerateTransposeForMult(A);CHKERRQ(ierr);
    cusparseMat->hasTranspose=PETSC_TRUE;
  }
  ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(yy,&yarray);CHKERRQ(ierr);
  try {
    ierr = cusparseMat->mat->multiplyTranspose(xarray, yarray);CHKERRCUSP(ierr);
  } catch (char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cusparseMat->mat->hasNonZeroStream()) {
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  }
  if (cusparseMat->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 3.0*cusparseMat->nonzerorow);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz - cusparseMat->nonzerorow);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMultAdd_SeqAIJCUSPARSE"
static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CUSPARRAY          *xarray,*yarray,*zarray;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  try {
    ierr = VecCopy_SeqCUSP(yy,zz);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(zz,&zarray);CHKERRQ(ierr);

    /* multiply add */
    ierr = cusparseMat->mat->multiplyAdd(xarray, zarray);CHKERRCUSP(ierr);

    ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);

  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  if (cusparseMat->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 2.0*cusparseMat->nonzerorow);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMultTransposeAdd_SeqAIJCUSPARSE"
static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CUSPARRAY          *xarray,*yarray,*zarray;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  if (!cusparseMat->hasTranspose) {
    ierr = MatSeqAIJCUSPARSEGenerateTransposeForMult(A);CHKERRQ(ierr);
    cusparseMat->hasTranspose=PETSC_TRUE;
  }
  try {
    ierr = VecCopy_SeqCUSP(yy,zz);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(zz,&zarray);CHKERRQ(ierr);

    /* multiply add with matrix transpose */
    ierr = cusparseMat->mat->multiplyAddTranspose(xarray, yarray);CHKERRCUSP(ierr);

    ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);

  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  if (cusparseMat->isSymmOrHerm) {
    ierr = PetscLogFlops(4.0*a->nz - 2.0*cusparseMat->nonzerorow);CHKERRQ(ierr);
  } else {
    ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_SeqAIJCUSPARSE"
static PetscErrorCode MatAssemblyEnd_SeqAIJCUSPARSE(Mat A,MatAssemblyType mode)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatAssemblyEnd_SeqAIJ(A,mode);CHKERRQ(ierr);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  }
  if (mode == MAT_FLUSH_ASSEMBLY) PetscFunctionReturn(0);
  A->ops->mult             = MatMult_SeqAIJCUSPARSE;
  A->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  A->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  A->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

/* --------------------------------------------------------------------------------*/
#undef __FUNCT__
#define __FUNCT__ "MatCreateSeqAIJCUSPARSE"
/*@
   MatCreateSeqAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format). This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective on MPI_Comm

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradgm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATSEQAIJCUSPARSE, MATAIJCUSPARSE
@*/
PetscErrorCode  MatCreateSeqAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt nz,const PetscInt nnz[],Mat *A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation_SeqAIJ(*A,nz,(PetscInt*)nnz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqAIJCUSPARSE"
static PetscErrorCode MatDestroy_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode     ierr;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  if (A->factortype==MAT_FACTOR_NONE) {
    try {
      if (A->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED) {
        delete (GPU_Matrix_Ifc*)(cusparseMat->mat);
      }
      if (cusparseMat->tempvec!=0) delete cusparseMat->tempvec;
      delete cusparseMat;
      A->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  } else {
    /* The triangular factors */
    try {
      Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
      GPU_Matrix_Ifc               *cusparseMatLo      = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
      GPU_Matrix_Ifc               *cusparseMatUp      = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
      delete (GPU_Matrix_Ifc*) cusparseMatLo;
      delete (GPU_Matrix_Ifc*) cusparseMatUp;
      delete (CUSPARRAY*) cusparseTriFactors->tempvec;
      delete cusparseTriFactors;
    } catch(char *ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  if (MAT_cusparseHandle) {
    hipsparseStatus_t stat;
    stat = hipsparseDestroy(MAT_cusparseHandle);CHKERRCUSP(stat);

    MAT_cusparseHandle=0;
  }
  /*this next line is because MatDestroy tries to PetscFree spptr if it is not zero, and PetscFree only works if the memory was allocated with PetscNew or PetscMalloc, which don't call the constructor */
  A->spptr = 0;

  ierr = MatDestroy_SeqAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqAIJCUSPARSE"
PETSC_EXTERN PetscErrorCode MatCreate_SeqAIJCUSPARSE(Mat B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate_SeqAIJ(B);CHKERRQ(ierr);
  if (B->factortype==MAT_FACTOR_NONE) {
    /* you cannot check the inode.use flag here since the matrix was just created.
       now build a GPU matrix data structure */
    B->spptr = new Mat_SeqAIJCUSPARSE;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->mat          = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->tempvec      = 0;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->format       = MAT_CUSPARSE_CSR;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->hasTranspose = PETSC_FALSE;
    ((Mat_SeqAIJCUSPARSE*)B->spptr)->isSymmOrHerm = PETSC_FALSE;
  } else {
    /* NEXT, set the pointers to the triangular factors */
    B->spptr = new Mat_SeqAIJCUSPARSETriFactors;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->loTriFactorPtr = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->upTriFactorPtr = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->tempvec        = 0;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->format         = cusparseMatSolveStorageFormat;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->hasTranspose   = PETSC_FALSE;
    ((Mat_SeqAIJCUSPARSETriFactors*)B->spptr)->isSymmOrHerm   = PETSC_FALSE;
  }
  /* Create a single instance of the MAT_cusparseHandle for any matrix (matMult, TriSolve, ...) */
  if (!MAT_cusparseHandle) {
    hipsparseStatus_t stat;
    stat = hipsparseCreate(&MAT_cusparseHandle);CHKERRCUSP(stat);
  }
  /* Here we overload MatGetFactor_petsc_C which enables -mat_type aijcusparse to use the
     default cusparse tri solve. Note the difference with the implementation in
     MatCreate_SeqAIJCUSP in ../seqcusp/aijcusp.cu */
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatGetFactor_petsc_C",MatGetFactor_seqaij_cusparse);CHKERRQ(ierr);

  B->ops->assemblyend      = MatAssemblyEnd_SeqAIJCUSPARSE;
  B->ops->destroy          = MatDestroy_SeqAIJCUSPARSE;
  B->ops->getvecs          = MatGetVecs_SeqAIJCUSPARSE;
  B->ops->setfromoptions   = MatSetFromOptions_SeqAIJCUSPARSE;
  B->ops->mult             = MatMult_SeqAIJCUSPARSE;
  B->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  B->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  B->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;

  ierr = PetscObjectComposeFunction((PetscObject)B, "MatCUSPARSESetFormat_C", MatCUSPARSESetFormat_SeqAIJCUSPARSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*M
   MATSEQAIJCUSPARSE - MATAIJCUSPARSE = "(seq)aijcusparse" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. All matrix calculations are performed on Nvidia GPUs using
   the CUSPARSE library. This type is only available when using the 'txpetscgpu' package.
   Use --download-txpetscgpu to build/install PETSc to use different CUSPARSE library and
   the different GPU storage formats.

   Options Database Keys:
+  -mat_type aijcusparse - sets the matrix type to "seqaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr - sets the storage format of matrices (for MatMult and factors in MatSolve) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid). Only available with 'txpetscgpu' package.
.  -mat_cusparse_mult_storage_format csr - sets the storage format of matrices (for MatMult) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid). Only available with 'txpetscgpu' package.
-  -mat_cusparse_solve_storage_format csr - sets the storage format matrices (for factors in MatSolve) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid). Only available with 'txpetscgpu' package.

  Level: beginner

.seealso: MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/
